// reference: https://github.com/srrcboy/dijkstra-CUDA/blob/master/dijkstra_cuda.cu

#include <hip/hip_runtime.h>
#include <stdio.h>  
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <iostream>
#include <unistd.h>
#include <fstream>

//Parameters; modify as needed
// #define VERTICES 16384           //number of vertices
// #define DENSITY 16              //minimum number of edges per vertex. DO NOT SET TO >= VERTICES
#define MAX_WEIGHT 100000      //max edge length + 1
#define INF_DIST 1000000000     //"infinity" initial value of each node
#define CPU_IMP 1               //number of Dijkstra implementations (non-GPU)
#define GPU_IMP 1               //number of Dijkstra implementations (GPU)
#define THREADS 2               //number of OMP threads
#define RAND_SEED 1234          //random seed
#define THREADS_BLOCK 32


int vertex_number;
float* dist;
int* visited;
int* parent;
float* graph;
int start;
std::string input_file;
std::string output_file;

int graph_size;
int int_array;
int data_array;


__global__ void closestNodeCUDA(float* node_dist, int* visited_node, int* global_closest, int num_vertices) {
    float dist = INF_DIST + 1;
    int node = -1;
    int i;

    for (i = 0; i < num_vertices; i++) {
        if ((node_dist[i] < dist) && (visited_node[i] != 1)) {
            dist = node_dist[i];
            node = i;
        }
    }

    global_closest[0] = node;
    visited_node[node] = 1;
}

__global__ void cudaRelax(float* graph, float* node_dist, int* parent_node, int* visited_node, int* global_closest, int num_vertices) {
    int next = blockIdx.x * blockDim.x + threadIdx.x;    //global ID
    int source = global_closest[0];

    float edge = graph[source * num_vertices + next];
    float new_dist = node_dist[source] + edge;

    if ((edge != 0) &&
        (visited_node[next] != 1) &&
        (new_dist < node_dist[next])) {
        node_dist[next] = new_dist;
        parent_node[next] = source;
    }

}

void setIntArrayValue(int* in_array, int array_size, int init_value) {
    int i;
    for (i = 0; i < array_size; i++) {
        in_array[i] = init_value;
    }
}

/*  Initialize elements of a 1D data_t array with an initial value   */
void setDataArrayValue(float* in_array, int array_size, float init_value) {
    int i;
    for (i = 0; i < array_size; i++) {
        in_array[i] = init_value;
    }
}

/*  Construct graph with no edges or weights     */
void initializeGraphZero(float* graph, int num_vertices) {
    int i, j;

    for (i = 0; i < num_vertices; i++) {
        for (j = 0; j < num_vertices; j++) {           //weight of all edges initialized to 0
            graph[i * num_vertices + j] = (float)0;
        }
    }
}

void dijkstra() {
    hipEvent_t exec_start, exec_stop;              //timer for execution only
    float elapsed_exec;                             //elapsed time
    hipEventCreate(&exec_start);
    hipEventCreate(&exec_stop);

    float* gpu_graph;
    float* gpu_dist;
    int* gpu_parent;
    int* gpu_visited;
    hipMalloc((void**)&gpu_graph, graph_size);
    hipMalloc((void**)&gpu_dist, data_array);
    hipMalloc((void**)&gpu_parent, int_array);
    hipMalloc((void**)&gpu_visited, int_array);

    int* closest = (int*)malloc(sizeof(int));
    *closest = -1;
    int* gpu_closest;
    hipMalloc((void**)&gpu_closest, sizeof(int));
    hipMemcpy(gpu_closest, closest, sizeof(int), hipMemcpyHostToDevice);

    setDataArrayValue(dist, vertex_number, INF_DIST);
    setIntArrayValue(parent, vertex_number, -1);
    setIntArrayValue(visited, vertex_number, 0);
    dist[start] = 0;

    hipMemcpy(gpu_graph, graph, graph_size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_dist, dist, data_array, hipMemcpyHostToDevice);
    hipMemcpy(gpu_parent, parent, int_array, hipMemcpyHostToDevice);
    hipMemcpy(gpu_visited, visited, int_array, hipMemcpyHostToDevice);

    dim3 gridClosest(1, 1, 1);
    dim3 blockClosest(1, 1, 1);

    dim3 gridRelax(vertex_number / THREADS_BLOCK, 1, 1);
    dim3 blockRelax(THREADS_BLOCK, 1, 1);

    hipEventRecord(exec_start);
    for (int i = 0; i < vertex_number; i++) {
        closestNodeCUDA<<<gridClosest, blockClosest>>>(gpu_dist, gpu_visited, gpu_closest, vertex_number);
        cudaRelax<<<gridRelax, blockRelax>>>(gpu_graph, gpu_dist, gpu_parent, gpu_visited, gpu_closest, vertex_number);
    }
    hipEventRecord(exec_stop);

    hipMemcpy(dist, gpu_dist, data_array, hipMemcpyDeviceToHost);
    hipMemcpy(parent, gpu_parent, int_array, hipMemcpyDeviceToHost);
    hipMemcpy(visited, gpu_visited, int_array, hipMemcpyDeviceToHost);

    hipFree(gpu_graph);
    hipFree(gpu_dist);
    hipFree(gpu_parent);
    hipFree(gpu_visited);

    hipEventElapsedTime(&elapsed_exec, exec_start, exec_stop);
    printf("\n\nCUDA Time (ms): %7.9f\n", elapsed_exec);
}

void contructGraph() {
    std::ifstream fin(input_file);
    fin >> vertex_number >> start; 
    printf("%s: vertex %d, start %d\n", input_file.c_str(), vertex_number, start);

    graph_size = vertex_number * vertex_number * sizeof(float);
    int_array = vertex_number * sizeof(int);
    data_array = vertex_number * sizeof(float);

    graph = (float*)malloc(graph_size);
    dist = (float*)malloc(data_array);
    parent = (int*)malloc(int_array);
    visited = (int*)malloc(int_array);

    for (int i = 0; i < vertex_number; i++) {
        for (int j = 0; j < vertex_number; j++) {
            fin >> graph[i * vertex_number + j];
        }
    }

    fin.close();
}

void clean() {
    free(graph);
    free(dist);
    free(parent);
    free(visited);
}

void write_graph() {
    std::ofstream out_file("test.txt", std::fstream::out);
    out_file << vertex_number << ' ' << start << '\n';

    for (int i = 0; i < vertex_number; i++) {
        for (int j = 0; j < vertex_number; j++) {
            float weight = graph[i * vertex_number + j];
            out_file << weight << ' ';
        }
        out_file << "\n";
    }

    out_file.close();
}

void write_output() {
    std::ofstream out_file(output_file, std::fstream::out);
    for (int i = 0; i < vertex_number; i++) {
        out_file << dist[i] << '\n';
    }

    out_file.close();
}

struct timespec diff(struct timespec start, struct timespec end)
{
    struct timespec temp;
    if ((end.tv_nsec - start.tv_nsec)<0) {
        temp.tv_sec = end.tv_sec - start.tv_sec - 1;
        temp.tv_nsec = 1000000000 + end.tv_nsec - start.tv_nsec;
    }
    else {
        temp.tv_sec = end.tv_sec - start.tv_sec;
        temp.tv_nsec = end.tv_nsec - start.tv_nsec;
    }
    return temp;
}

int main(int argc, char *argv[]) {

    int opt;

    if (argc < 3) {
        std::cerr << "Usage: " << argv[0] << " -f input_filename -o output_filename\n";
        exit(1);
    }

    while ((opt = getopt(argc, argv, "f:o:")) != -1) {
        switch (opt) {
        case 'f':
            input_file = optarg;
            break;
        case 'o':
            output_file = optarg;
            break;
        default:
            std::cerr << "Usage: " << argv[0] << " -f input_filename -o output_filename\n";
        }
    }

    contructGraph();
    dijkstra();
    write_output();
    clean();
    return 0;
}